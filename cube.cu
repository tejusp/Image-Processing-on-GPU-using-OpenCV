#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <device_functions.hpp>
#include <>
#include <hip/driver_types.h>
#include <host_defines.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/core/gpumat.hpp>
#include <opencv2/gpu/device/common.hpp>
#include <stdio.h>
#include <hip/hip_vector_types.h>
#include <ostream>
#include <string>
#include <iostream>
#include <typeinfo>
using namespace std;
using std::cout;

#define arraySIZE 240

__shared__ int edgesValues[10][24];
__shared__ int counter;
__constant__ const int maxContourPoints = 30;

__global__ void funcKernel(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("test=%d", edgesValues[0]);

	if (rowInd >= rows || colInd >= cols)
		return;
	const float* rowsrcptr = (const float *) (((char *) srcptr)
			+ rowInd * srcstep);
//	float* rowdstPtr = (float *) (((char *) dstptr) + rowInd * dststep);
	float val = rowsrcptr[colInd];
//	printf("test");
//	printf("\nat row=%d col=%d inp array=%d ", rowInd, colInd,
//			inputArray_d[rowInd * cols + colInd]);

	if ((rowInd > 2 && rowInd < (rows - 2))
			&& (colInd > 2 && colInd < (cols - 2))) {
		if (val == 255) {
			const float* rowsrcptrNxt = (const float *) (((char *) srcptr)
					+ (rowInd + 1) * srcstep);
			const float* rowsrcptrPrev = (const float *) (((char *) srcptr)
					+ (rowInd - 1) * srcstep);
			if (rowsrcptrPrev[colInd - 1] == 0 || rowsrcptrPrev[colInd] == 0
					|| rowsrcptrPrev[colInd + 1] == 0
					|| rowsrcptr[colInd - 1] == 0 || rowsrcptr[colInd - 1] == 0
					|| rowsrcptrNxt[colInd - 1] == 0
					|| rowsrcptrNxt[colInd] == 0
					|| rowsrcptrNxt[colInd + 1] == 0) {

				edgesValues[rowInd][colInd] = 1;
				//outputArray_d[rowInd * cols + colInd] = 1;
//				edgesValues[rowInd * cols + colInd] = 1;
//				printf("\nat row=%d col=%d out araay=%d ", rowInd, colInd,
//						outputArray_d[rowInd * cols + colInd]);

//				printf(
//						"\nat row=%d col=%d ;val=%f, rowsrcptr[colInd-1]=%f, rowsrcptr[colInd+1]=%f,rowsrcptrNxt =%f",
//						rowInd, colInd, val, rowsrcptr[colInd - 1],
//						rowsrcptr[colInd + 1], rowsrcptrNxt[colInd]);

			} else {
				edgesValues[rowInd][colInd] = 0;
//				edgesValues[rowInd * cols + colInd] = 0;
//
//				outputArray_d[rowInd * cols + colInd] = inputArray_d[rowInd
//						* cols + colInd];
			}

		}
	}

	for (int i = 0; i < rows * cols; i++) {
//		printf("in loop=%d", i);
	}

}

__global__ void funcKernel2(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowInd >= rows || colInd >= cols)
		return;
//	const float* rowsrcptr = (const float *) (((char *) srcptr)
//			+ rowInd * srcstep);
//	float* rowdstPtr = (float *) (((char *) dstptr) + rowInd * dststep);
//	printf("\nat row=%d col=%d inp array=%d ", rowInd, colInd,
//			inputArray_d[rowInd * cols + colInd]);
//	__shared__ int test[240];
	int contourVal[maxContourPoints][2];
	counter = 0;
	int maxRow = 0;
	int minRow = rows;
	int minCol = cols;
	int maxCol = 0;
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if (edgesValues[i][j] == 1 && counter < maxContourPoints) {
				if (i < minRow) {
					minRow = i;
				}
				if (i > maxRow) {
					maxRow = i;
				}
				if (j < minCol) {
					minCol = j;
				}
				if (j > maxCol) {
					maxCol = j;
				}
				contourVal[counter][1] = i;
				contourVal[counter][2] = j;
				printf("%d - test contour at %d,%d is %d \n", counter, i, j,
						edgesValues[i][j]);
				counter++;

			}
		}
	}

	int centroidRow = (minRow + maxRow) / 2;
	int centroidCol = (minCol + maxCol) / 2;
	printf(
			"minRow=%d,maxRow=%d,minCol=%d,maxCol=%d,centroidRow=%d,centroidCol=%d",
			minRow, maxRow, minCol, maxCol, centroidRow, centroidCol);

}

__global__ void funcKernel3(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowInd >= rows || colInd >= cols)
		return;
//	const float* rowsrcptr = (const float *) (((char *) srcptr)
//			+ rowInd * srcstep);
//	float* rowdstPtr = (float *) (((char *) dstptr) + rowInd * dststep);
//	printf("\nat row=%d col=%d inp array=%d ", rowInd, colInd,
//			inputArray_d[rowInd * cols + colInd]);
//	__shared__ int test[240];

	if (counter < 50) {
		__shared__ int contourPoints[20];
	} else if (counter < 100) {
		__shared__ int contourPoints[100];
	} else if (counter < 150) {
		__shared__ int contourPoints[20];
	} else if (counter < 200) {
		__shared__ int contourPoints[100];
	}
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if (edgesValues[i][j] == 1) {
				counter++;
				printf("%d - test contour at %d,%d is %d \n", counter, i, j,
						edgesValues[i][j]);
			}
		}
	}

}

int divUp(int a, int b) {
	return (a + b - 1) / b;
}

//extern "C"
//{
void func(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows) {
	dim3 blDim(32, 8);
	dim3 grDim(divUp(cols, blDim.x), divUp(rows, blDim.y));
//	size_t size = sizeof(int);

	int inputArray_h[rows * cols];
	int outputArray_h[rows * cols];
	int* inputArray_d;
	int* outputArray_d;

	for (int j = 0; j < rows * cols; j++) {
		inputArray_h[j] = 0;
	}

//	for (int i = rows * cols - 1; i >= 0; i--)
//		cout << "==" << inputArray_h[i];

	int ARRAY_BYTES = rows * cols * sizeof(int);
//allocate GPU memory

	hipMalloc((void**) &inputArray_d, ARRAY_BYTES);
	hipMalloc((void**) &outputArray_d, ARRAY_BYTES);

//	hipMalloc((void**) &inputMatrix_d, ARRAY_BYTES);
//	hipMalloc((void**) &outputMatrix_d, ARRAY_BYTES);

	hipMemcpy(inputArray_d, inputArray_h, ARRAY_BYTES, hipMemcpyHostToDevice);

	std::cout << "calling kernel from func\n";
	funcKernel<<<grDim, blDim>>>(srcptr, dstptr, srcstep, dststep, cols, rows,
			inputArray_d, outputArray_d);
	hipDeviceSynchronize();
	funcKernel2<<<1, 1>>>(srcptr, dstptr, srcstep, dststep, cols, rows,
			inputArray_d, outputArray_d);

	hipMemcpy(outputArray_d, outputArray_h, ARRAY_BYTES,
			hipMemcpyDeviceToHost);

//	if (edgesValues[0]) {
//		cout << "host: " << edgesValues[0] << endl;
//	}

	cout << "\n\nstarting output in host" << endl;

//	for (int i = rows * cols - 1; i >= 0; i--)
//		cout << "==" << (int) outputArray_h[i];

//	int *test;
//	hipMemcpy(counter, test, sizeof(int));
	hipDeviceSynchronize();
//	std::cout << "done with kernel call\n==" << counter << endl;
}
//}
