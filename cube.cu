#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>
#include <host_defines.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/gpumat.hpp>
#include <opencv2/core/mat.hpp>
#include <opencv2/core/operations.hpp>
#include <opencv2/core/types_c.h>
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/highgui/highgui_c.h>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/imgproc/types_c.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <fstream>
#include <iosfwd>
#include <ostream>
#include <sstream>
#include <string>
#include <vector>
#include <iostream>

#include "csvparser.h"

using namespace std;

using namespace cv;
using namespace cv::gpu;

struct Arguments {
	string project;
	string input;
	string output;
	int padding;
	int frames;
	string extension;
	int width;
	int height;
	int area_min;
	int area_max;
	int search_win_size;
	int blur_radius;
	int threshold_win_size;
	float threshold_ratio;
	string log;
	bool verbose;

	Arguments() :
			input("data/"), output("output.txt"), padding(7), frames(1000), extension(
					".jpg"), width(640), height(480), area_min(200), area_max(
					400), search_win_size(100), blur_radius(3), threshold_win_size(
					25), threshold_ratio(0.9), log("wormSeg.log"), verbose(true) {
	}
} cla;

int findCentroidFrom1Image(cv::Mat, int*, int*, int*);

template<typename T> string NumberToString(T pNumber) {
	ostringstream oOStrStream;
	oOStrStream << pNumber;

	return oOStrStream.str();
}

string intToFileName(string fileNameFormat, int fileNumber) {
	string temp = NumberToString(fileNumber);

	return fileNameFormat.replace(fileNameFormat.size() - temp.size(),
			temp.size(), temp);
}
void func(const float*, float*, size_t, const size_t, int, int, int&, int&);
int centroidRow = 0;
int centroidCol = 0;

void callKernel(const cv::gpu::GpuMat &src, cv::gpu::GpuMat &dst, int *pX,
		int *pY) {
	float* p = (float*) src.data;
	float* p2 = (float*) dst.data;
	func(p, p2, src.step, dst.step, src.cols, src.rows, centroidRow,
			centroidCol);
//	*pX = centroidRow;
//	*pY = centroidCol;
//	cout << "print row=" << *pX << endl;
//	cout << "print col=" << *pY << endl;
}

int cudaFindCentroid(cv::Mat src, int *pX, int *pY, int *pArea) {
	//GPU Mat... Copy from CPU memory to GPU memory...
	cv::gpu::GpuMat gpu_src(src);

	cv::gpu::GpuMat matAfterBlur;
	//Filters on GPU...
	cv::gpu::blur(gpu_src, matAfterBlur,
			Size(cla.blur_radius, cla.blur_radius));
	cv::gpu::GpuMat matAfterThreshold;
	//Convert into Binary image on GPU...
	cv::gpu::threshold(matAfterBlur, matAfterThreshold,
			int(cla.threshold_ratio * 255), 255, THRESH_BINARY_INV);

	cv::gpu::GpuMat floatMatForKernel;
	matAfterThreshold.convertTo(floatMatForKernel, CV_32FC1);
//	cout << "i'm here" << endl;

	callKernel(floatMatForKernel, gpu_src, pX, pY);
//	//Copy from GPU memory to CPU memory...
//	if (*pX) {
////		*pX = bRect.x + (bRect.width / 2);
////		*pY = bRect.y + (bRect.height / 2);
//		*pArea = 10;
//	} else {
//		*pX = -1;
//		*pY = -1;
//		*pArea = -1;
//	}
//
//	return 0;
}

int wormSegmenter() {

	fstream outputFile;

	outputFile.open(cla.output.c_str(), ios::out);

	int x = -1, y = -1, area = -1;
	int adjustX = 0, adjustY = 0;

	for (int fileNumber = 0; fileNumber < cla.frames; fileNumber++) {
		string fileName = cla.input + intToFileName("0000000", fileNumber)
				+ cla.extension;
		cv::Mat src = cv::imread(fileName, CV_LOAD_IMAGE_GRAYSCALE);

		if (!src.data) {
//			cout << endl << "Exited." << endl;
			exit(1);
		}

		if ((x == -1) && (y == -1)) {
			findCentroidFrom1Image(src, &x, &y, &area);
			src = cv::imread(fileName, CV_LOAD_IMAGE_GRAYSCALE);
			adjustX = x - (cla.search_win_size / 2);
			adjustY = y - (cla.search_win_size / 2);
		} else {
			src = src(
					cv::Rect(x - (cla.search_win_size / 2),
							y - (cla.search_win_size / 2), cla.search_win_size,
							cla.search_win_size));
			cudaFindCentroid(src, &x, &y, &area);
			if ((x > 0) && (y > 0)) {

				//std::cout << "writing file=" << fileNumber << "x=" << x << "y="					<< y << endl;

//				x += adjustX;
//				y += adjustY;
//				adjustX = x - (cla.search_win_size / 2);
//				adjustY = y - (cla.search_win_size / 2);
				x = 153;
				y = 251;
			}
		}
//		cout << "writing file=" << fileNumber << "x=" << x << "y=" << y << endl;
		outputFile << fileNumber << ", " << x << ", " << y << ", " << area
				<< endl;
	}

	outputFile.close();

	return 0;
}

int findCentroidFrom1Image(cv::Mat src, int *pX, int *pY, int *pArea) {
	// Smoothing the image.
	blur(src, src, Size(cla.blur_radius, cla.blur_radius)); //Blur radius 3 in original java worm segmenter.

	// Convert the image into binary image.
	threshold(src, src, int(cla.threshold_ratio * 255), 255, THRESH_BINARY_INV);

	// Vector for storing contour
	vector<vector<Point> > contours;

	vector<Vec4i> hierarchy;

	// Find contours in the image.
	findContours(src, contours, hierarchy, CV_RETR_CCOMP,
			CV_CHAIN_APPROX_SIMPLE);

	if (contours.size() > 0) {
		int largest_contour_index = 0;
		int largest_area = 0;

		// Iterate through each contour.
		for (int i = 0; i < contours.size(); i++) {
			//  Find the area of contour
			double a = contourArea(contours[i], false);

			if (a > largest_area) {
				largest_area = a;

				// Store the index of largest contour
				largest_contour_index = i;
			}
		}

		Rect bRect = boundingRect(contours[largest_contour_index]);

		*pX = bRect.x + (bRect.width / 2);
		*pY = bRect.y + (bRect.height / 2);
		*pArea = largest_area;
	} else {
		*pX = -1;
		*pY = -1;
		*pArea = -1;
	}

	return 0;
}

int main(int argc, char **argv) {
	int i = 0;
	//                                   file, delimiter, first_line_is_header?
	CsvParser *csvparser = CsvParser_new("example_file.csv", ",", 0);
	CsvRow *row;
	std::vector<std::vector<float> > vec;
	while ((row = CsvParser_getRow(csvparser))) {
		std::vector<float> eachLine;
		//printf("==NEW LINE==\n");
		const char **rowFields = CsvParser_getFields(row);
		for (i = 0; i < CsvParser_getNumFields(row); i++) {
			eachLine.push_back(atof(rowFields[i]));
			//printf("FIELD: %f\n", eachLine[i]);
		}
		vec.push_back(eachLine);
//		printf("\n");
		CsvParser_destroy_row(row);
	}
//	printf("test=%f\n", vec[0][1]);
//	printf("test=%f\n", vec[1][0]);
//	printf("test=%f\n", vec[2][0]);
//	printf("test=%f\n", vec[3][0]);
	CsvParser_destroy(csvparser);

	int a = wormSegmenter();
	return 0;
}

//#define arraySIZE 240
__device__ int edgesValues[100][100];
//__device__ int edgesValues[480][640];
__shared__ int counter;
__constant__ const int maxContourPoints = 300;

__global__ void funcKernel(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;

	if (rowInd >= rows || colInd >= cols)
		return;
	const float* rowsrcptr = (const float *) (((char *) srcptr)
			+ rowInd * srcstep);
	float val = rowsrcptr[colInd];

	if ((rowInd > 2 && rowInd < (rows - 2))
			&& (colInd > 2 && colInd < (cols - 2))) {
		if (val == 255) {
			const float* rowsrcptrNxt = (const float *) (((char *) srcptr)
					+ (rowInd + 1) * srcstep);
			const float* rowsrcptrPrev = (const float *) (((char *) srcptr)
					+ (rowInd - 1) * srcstep);
			if (rowsrcptrPrev[colInd - 1] == 0 || rowsrcptrPrev[colInd] == 0
					|| rowsrcptrPrev[colInd + 1] == 0
					|| rowsrcptr[colInd - 1] == 0 || rowsrcptr[colInd - 1] == 0
					|| rowsrcptrNxt[colInd - 1] == 0
					|| rowsrcptrNxt[colInd] == 0
					|| rowsrcptrNxt[colInd + 1] == 0) {

				edgesValues[rowInd][colInd] = 1;
			} else {
				edgesValues[rowInd][colInd] = 0;
			}

		}
	}

}

__global__ void funcKernel2(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d, int *a, int *b, int *c) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowInd >= rows || colInd >= cols)
		return;

	counter = 0;
	int maxRow = 0;
	int minRow = rows;
	int minCol = cols;
	int maxCol = 0;
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if (edgesValues[i][j] == 1 && counter < maxContourPoints) {
				if (i < minRow) {
					minRow = i;
				}
				if (i > maxRow) {
					maxRow = i;
				}
				if (j < minCol) {
					minCol = j;
				}
				if (j > maxCol) {
					maxCol = j;
				}
				counter++;

			}
		}
	}

	int centroidRow = (minRow + maxRow) / 2;
	int centroidCol = (minCol + maxCol) / 2;
//	printf("%d,%d", centroidRow, centroidCol);
	*a = centroidRow;
	*b = centroidCol;
	*c = *a + *b;

}

int divUp(int a, unsigned int b) {
	return (a + b - 1) / b;
}

void func(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int& centroidRow,
		int& centroidCol) {
	dim3 blDim(32, 8);
	dim3 grDim(divUp(cols, blDim.x), divUp(rows, blDim.y));

	int inputArray_h[rows * cols];
	int outputArray_h[rows * cols];
	int* cRowNumber;
	int* cColNumber;

	for (int j = 0; j < rows * cols; j++) {
		inputArray_h[j] = 0;
	}

	int int_BYTES = sizeof(int);
	//allocate GPU memory

	hipMalloc((void**) &cRowNumber, int_BYTES);
	hipMalloc((void**) &cColNumber, int_BYTES);

	hipMemcpy(cRowNumber, inputArray_h, int_BYTES, hipMemcpyHostToDevice);

	funcKernel<<<grDim, blDim>>>(srcptr, dstptr, srcstep, dststep, cols, rows,
			cRowNumber, cColNumber);
//	hipDeviceSynchronize();

	int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
	int size = sizeof(int); // we need space for an integer
	// allocate device copies of a, b, c
	hipMalloc((void**) &dev_a, size);
	hipMalloc((void**) &dev_b, size);
	hipMalloc((void**) &dev_c, size);
	// copy inputs to device

	funcKernel2<<<1, 1>>>(srcptr, dstptr, srcstep, dststep, cols, rows,
			cRowNumber, cColNumber, dev_a, dev_b, dev_c);

}
